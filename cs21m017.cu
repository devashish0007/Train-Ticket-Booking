#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>

#define M 25
#define N 100000
#define BLOCKSIZE 1024

using namespace std;


__global__ void init(int * lock, int size)
{
	unsigned id =  blockIdx.x * BLOCKSIZE + threadIdx.x;
	if(id < size)
		lock[id] = 10000;
	
}

__global__ void reserve(int R, int * d_lock, int *d_T, int *d_C, int *d_S, int *d_D, int *d_size, int *d_offset, int volatile *d_seat, int *d_capacity, int * d_class_offset, int *d_R_T, int *d_R_C, int *d_R_S, int *d_R_D, int *d_R_size, int *d_R_seat, int *d_R_result, int start)
{

	int id =  start + blockIdx.x * BLOCKSIZE + threadIdx.x;
	int destination, source, size, train, flag, seat_offset, count, select, execute;
	__shared__ unsigned complete;//, lockvar, round;
//	round = 0;
//	lockvar=0;
	complete = 1;
		
	
	train = d_R_T[id];
	//class_n = d_offset[train] + d_R_C[id];
	seat_offset = d_offset[train] + d_R_C[id] * d_size[train];
	select = d_class_offset[train]+d_R_C[id];
	execute = 1;
	
	flag = 0;
	
	__syncthreads(); // subject to cause deadlock
		
	if(d_R_S[id] > d_R_D[id])
	{
		destination = abs(d_R_S[id] - d_D[train]);
		source = abs(d_R_D[id] - d_D[train]);
		size = abs(source - destination);						    			
	}
	else 
	{
		source = abs(d_R_S[id] - d_S[train]);
		destination = abs(d_R_D[id] - d_S[train]);
		size = abs(source - destination);
	}

		

	/**************************reservation starts*****************************/
		
	do
	{
		complete = 1;
//		if(id ==3)
	//		printf("Round %d\n", round++);
				
		if(execute)
		{
			atomicMin(&(d_lock[select]),id);
		}
		__syncthreads();

		
		if(execute)
		{			
		
			//printf("%d Lock acquired by :%d\n", id);
			if(d_lock[select] == id)
			{
				//printf("Lock acquired by :%d -> %d\n", id,  train);
				// Allot seat
				for(int i = seat_offset + source; i < (seat_offset + destination); i++)
				{
					d_seat[i] = d_seat[i] - d_R_seat[id];
					
					if(d_seat[i] < 0)
					{	
						flag = 1;
						count = i;
						break;						
					}
				}
				
				if(flag)
				{
					// if seat allocation fails
					for(int i = (seat_offset + source); i <= count; i++)
					{
						d_seat[i] += d_R_seat[id];
					}
					d_R_result[id] = 0;
					flag = 0;
				}
				else
				{			
					// if successfull store the result
					d_R_result[id] = size * d_R_seat[id];
				}
				execute = 0;
				//printf("Lock released by :%d", id);
				// unlock the class
				d_lock[select] = 10000;	
				//printf("Lock released by :%d -> %d : %d\n", id,  train, d_R_C[id]);	
			
			}
			else if(execute)
			{
				complete = 0;
			}	
		}
		__syncthreads();
				
	}while(!complete);
	
	__syncthreads();
	/*******************Reservation ends**************************/

}



int main()
{
	int n, count=0, seat_count=0;
	cin >> n;
	
// Allocate memory on cpu
	
    int *T = (int *) malloc ( (n) * sizeof (int) );
    int *C = (int *) malloc ( (n) * sizeof (int) );
    int *S = (int *) malloc ( (n) * sizeof (int) );
    int *D = (int *) malloc ( (n) * sizeof (int) );
    int *size = (int *) malloc ( (n) * sizeof (int) );
    int *offset = (int *) malloc ( (n) * sizeof (int) );
    int *seat = (int *) malloc ( (M * n * 50) * sizeof (int) );
    int *capacity = (int *) malloc ( (M * n) * sizeof (int) ); 
    int * class_offset = (int *) malloc ( (n) * sizeof (int) ); 
      
// Allocate memory on gpu	
	int *d_T, *d_C, *d_S, *d_D, *d_size, *d_offset, *d_seat, *d_capacity, *d_class_offset;

	hipMalloc(&d_T, (n) * sizeof(int));
	hipMalloc(&d_C, (n) * sizeof(int));
	hipMalloc(&d_S, (n) * sizeof(int));
	hipMalloc(&d_D, (n) * sizeof(int));
	hipMalloc(&d_size, (n) * sizeof(int));
	hipMalloc(&d_offset, (n) * sizeof(int));	
	hipMalloc(&d_capacity, (M * n) * sizeof(int));
	hipMalloc(&d_class_offset, (M * n) * sizeof(int));
	hipMalloc(&d_seat, (M * n * 50) * sizeof(int));
	
	for(int i = 0; i < n; i++)
	{
		int t,s,d,c;
		cin >> t >> c >> s >> d;
		T[i] = t;
		S[i] = s;
		D[i] = d;
		C[i] = c;
		offset[i] = seat_count;
		class_offset[i] = count;
		size[i] = abs(s - d);
		for(int j = 0; j < c; j++)
		{
			int c_no, maxc;

			cin >> c_no >> maxc;
			for(int k=seat_count; k < (seat_count + size[i]); k++)
				seat[k] = maxc;
			seat_count += size[i];
			capacity[count] = maxc;
			count++;
		}
	}

	// Copy memory from host to device
	hipMemcpy(d_T, T, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_S, S, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_D, D, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_size, size, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_offset, offset, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_capacity, capacity, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_seat, seat, seat_count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_class_offset, class_offset, seat_count * sizeof(int), hipMemcpyHostToDevice);
	
	// setup lock array	
	
	int *d_lock;
	hipMalloc(&d_lock, (count) * sizeof(int));
	int initblock = ceil((float(count)/BLOCKSIZE));
	init<<<initblock,BLOCKSIZE>>>(d_lock, count);	


	// Take request input for reservation


	int B, R;
	cin >> B;
	// Allocate memory on cpu
	for(int i = 0; i < B; i++)
	{
		cin>> R;
		
		int *R_T = (int *) malloc ( (R) * sizeof (int) );
		int *R_C = (int *) malloc ( (R) * sizeof (int) );
		int *R_S = (int *) malloc ( (R) * sizeof (int) );
		int *R_D = (int *) malloc ( (R) * sizeof (int) );
		int *R_seat = (int *) malloc ( (R) * sizeof (int) );
		int *R_result = (int *) malloc ( (R) * sizeof (int) );
		int *R_size = (int *) malloc ( (R) * sizeof (int) );

		  
		// Allocate memory on gpu	
		int *d_R_T, *d_R_C, *d_R_S, *d_R_D, *d_R_size, *d_R_seat, *d_R_result, id;

		hipMalloc(&d_R_T, (R) * sizeof(int));
		hipMalloc(&d_R_C, (R) * sizeof(int));
		hipMalloc(&d_R_S, (R) * sizeof(int));
		hipMalloc(&d_R_D, (R) * sizeof(int));
		hipMalloc(&d_R_seat, (R) * sizeof(int));
		hipMalloc(&d_R_result, (R) * sizeof(int));
		hipMalloc(&d_R_size, (R) * sizeof(int));	

		for(int j = 0; j < R; j++)
		{
			cin>> id>> R_T[j] >> R_C[j] >> R_S[j] >> R_D[j] >> R_seat[j];
			R_size[j] = abs(R_S[j] - R_D[j]);	
		}
		
		// Copy memory from host to device
		hipMemcpy(d_R_T, R_T, R * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_R_C, R_C, R * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_R_S, R_S, R * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_R_D, R_D, R * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_R_seat, R_seat, R * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_R_size, R_size, R * sizeof(int), hipMemcpyHostToDevice);
		
		// Organize request.
		int nblocks = R / BLOCKSIZE;
		int extra = R % BLOCKSIZE;
		
		int k = nblocks;
		// Make reservations.
		while(k)
		{	

			reserve<<<1,BLOCKSIZE>>>(R, d_lock, d_T, d_C, d_S, d_D, d_size, d_offset, d_seat, d_capacity, d_class_offset, d_R_T, d_R_C, d_R_S, d_R_D, d_R_size, d_R_seat, d_R_result, BLOCKSIZE*(nblocks - k));
			hipDeviceSynchronize();
			k--;
		}
		
		reserve<<<1,extra>>>(R, d_lock, d_T, d_C, d_S, d_D, d_size, d_offset, d_seat, d_capacity, d_class_offset, d_R_T, d_R_C, d_R_S, d_R_D, d_R_size, d_R_seat, d_R_result, BLOCKSIZE*nblocks);


		// copy the result back...
		hipMemcpy(R_result, d_R_result, R * sizeof(int), hipMemcpyDeviceToHost);
		
		// print result
		long success=0, fail=0, total_seat=0;
		for(int j = 0; j < R; j++)
		{
			if(R_result[j])
			{
				cout<<"success\n";
				success++;
				total_seat += R_result[j];
			}
			else
			{
				cout<<"failure\n";
				fail++;
			}
		}
		
		cout<< success<<" " << fail<<"\n";
		cout<< total_seat <<"\n";
		total_seat = 0;
		free(R_T);
		free(R_C);
		free(R_S);
		free(R_D);
		free(R_seat);
		free(R_result);
	}
	
hipError_t err = hipGetLastError();
//printf("error=%d, %s, %s\n", err, cudaGetErrorName(err), cudaGetErrorString(err));

	
	return 0;
}
